/* -------------------------------------------------------------------------- */
// Failed attempt to find monoliths with cuda.
//
// I'm somewhat sure the lack of memory locality between different seeds and
// the large-ish fractal perlin structs for the tiny cores killed performance.
//
// Maybe it's possible to fix it, or perhaps certain computations are just
// better suited for the CPU even if embarrassingly parallel. This was my first
// time toying with cuda, chances are I did something wrong, PRs are welcome
// to improve this code for the endeavor to find the largest monoliths :^)
//
// Compiling and running:
// - Install CUDA from your package manager (nvcc), have it on path, good luck
//
// Either manually:
//   - Run: `meson setup --buildtype release ./build --reconfigure`
//   - Run: `ninja -C ./build`
//   - Run: `./build/monolithium`
//
// Or install [uv](https://docs.astral.sh/uv/):
//   - Run: `uv run monolithium-cuda`
/* -------------------------------------------------------------------------- */

#include <cmath>
#include <cstdint>
#include <cstdio>
#include <unistd.h>

#include <hip/hip_runtime.h>

#define Gpu __device__
#define Cpu __host__

#define SKIP_REJECTION 1
#define SKIP_TABLE 1

/* -------------------------------------------------------------------------- */
// Utility functions

Gpu inline float fade(float t) {
    return t * t * t * (t * (t * 6.0 - 15.0) + 10.0);
}

Gpu inline float lerp(float t, float a, float b) {
    // return a + t * (b - a);
    return fma(t, b - a, a);
}

Gpu inline float grad(uint8_t hash, float x, float y, float z) {
    int h = hash & 15;
    float u = h < 8 ? x : y;
    float v = h < 4 ? y : h == 12 || h == 14 ? x : z;
    return ((h & 1) == 0 ? u : -u) + ((h & 2) == 0 ? v : -v);
}

/* -------------------------------------------------------------------------- */
// Java RNG implementation

constexpr int64_t M = (1LL << 48) - 1;
constexpr int64_t A = 0x5DEECE66DLL;
constexpr int64_t C = 11LL;

constexpr double F64_DIV = (1ULL << 53);

struct JavaRNG {
    int64_t state;

    Gpu inline JavaRNG(uint64_t seed) {
        this->state = ((int64_t) seed ^ A) & M;
    }

    /// Roll the state, same effect as ignoring a `.next()` call
    Gpu inline void step() {
        this->state = (this->state * A + C) & M;
    }

    /// Rolls the state and returns N low bits
    Gpu inline int32_t next(uint8_t bits) {
        this->step();
        return (int32_t) (this->state >> (48 - bits));
    }

    Gpu inline int32_t next_i32_bound(int32_t max) {
        if (__popc(max) == 1) {
            return (int32_t)(((int64_t) max * (int64_t) this->next(31)) >> 31);
        } else {
            int32_t next = this->next(31);
            int32_t take = next % max;

            #if SKIP_REJECTION
            #else
                while (next - take + max - 1 < 0) {
                    next = this->next(31);
                    take = next % max;
                }
            #endif

            return take;
        }
    }

    Gpu inline double next_f64() {
        int64_t high = ((int64_t) this->next(26)) << 27;
        int64_t low  =  (int64_t) this->next(27);
        return (double)(high | low) / F64_DIV;
    }
};

/* -------------------------------------------------------------------------- */

struct PerlinNoise {
    uint8_t map[256];
    float xoff;
    float yoff;
    float zoff;

    Gpu void init(JavaRNG* rng) {
        this->xoff = (float) (rng->next_f64() * 256.0);
        this->yoff = (float) (rng->next_f64() * 256.0);
        this->zoff = (float) (rng->next_f64() * 256.0);

        // Start a new 'arange' array
        for (int i=0; i<256; i++) {
            this->map[i] = i & 0xFF;
        }

        // Shuffle the first half
        for (int a=0; a<256; a++) {
            int b = a + rng->next_i32_bound(256 - a);
            uint8_t temp = this->map[a];
            this->map[a] = this->map[b];
            this->map[b] = temp;
        }
    }

    Gpu float get_map(int index) {
        return this->map[index & 0xFF];
    }

    /// Sample the noise at a given coordinate
    /// - Note: For monoliths, y is often 0.0
    Gpu float sample(float x, float y, float z) {
        x += this->xoff;
        y += this->yoff;
        z += this->zoff;

        // Convert to grid coordinates (512 length)
        int xi = __float2int_rd(x) & 0xFF;
        int yi = __float2int_rd(y) & 0xFF;
        int zi = __float2int_rd(z) & 0xFF;

        // Get the fractional parts
        float xf = x - floor(x);
        float yf = y - floor(y);
        float zf = z - floor(z);

        // Smoothstep-like factors
        float u = fade(xf);
        float v = fade(yf);
        float w = fade(zf);

        // Get the hash values for the corners
        int a  = this->get_map(xi + 0 + 0);
        int aa = this->get_map(yi + a + 0);
        int ab = this->get_map(yi + a + 1);
        int b  = this->get_map(xi + 0 + 1);
        int ba = this->get_map(yi + b + 0);
        int bb = this->get_map(yi + b + 1);

        return lerp(w,
            lerp(v,
                lerp(u, grad(this->get_map(aa + zi), xf, yf, zf),
                        grad(this->get_map(ba + zi), xf - 1.0, yf, zf)),
                lerp(u, grad(this->get_map(ab + zi), xf, yf - 1.0, zf),
                        grad(this->get_map(bb + zi), xf - 1.0, yf - 1.0, zf))),
            lerp(v,
                lerp(u, grad(this->get_map(aa + zi + 1), xf, yf, zf - 1.0),
                        grad(this->get_map(ba + zi + 1), xf - 1.0, yf, zf - 1.0)),
                lerp(u, grad(this->get_map(ab + zi + 1), xf, yf - 1.0, zf - 1.0),
                        grad(this->get_map(bb + zi + 1), xf - 1.0, yf - 1.0, zf - 1.0))));
    }

    /// Roll the generator state that would have created a PerlinNoise
    /// - Fast way around without as many memory operations
    Gpu static void discard(JavaRNG* rng, int count) {

        // Gotta love magic numbers!
        #if SKIP_TABLE
            rng->state *= 249870891710593LL;
            rng->state += 44331453843488LL;
            rng->state &= M;
            return;
        #endif

        for (int i=0; i<count; i++) {

            // Coordinates f64 offsets
            for (int j=0; j<3; j++) {
                rng->next_f64();
            }

            // Permutations swapping
            for (int max=256; max>=1; max--) {
                rng->next_i32_bound(max);
            }
        }
    }
};

/* -------------------------------------------------------------------------- */

template<int OCTAVES> struct FractalPerlin {
    PerlinNoise noise[OCTAVES];

    Gpu void init(JavaRNG* rng) {
        for (int i=0; i<OCTAVES; i++) {
            this->noise[i].init(rng);
        }
    }

    Gpu float sample(float x, float y, float z) {
        float sum = 0.0f;
        for (int i=0; i<OCTAVES; i++) {
            int   j = OCTAVES - 1 - i;
            float s = (float) (1 << j);
            sum += this->noise[j].sample(x/s, y/s, z/s) * s;
        }
        return sum;
    }
};

/* -------------------------------------------------------------------------- */

struct World {
    FractalPerlin<10> hill;
    FractalPerlin<16> depth;

    Gpu void init(uint64_t seed) {
        JavaRNG rng(seed);

        // Skip 48 generators priorly used elsewhere
        PerlinNoise::discard(&rng, 48);

        this->hill.init(&rng);
        this->depth.init(&rng);
    }

    // Check if a given coordinate is part of a monolith
    Gpu bool is_monolith(int64_t x, int64_t z) {
        float depth = this->depth.sample(
            (float) (x/4) * 100.0, 0.0,
            (float) (z/4) * 100.0
        );

        if (fabs(depth) < 8000.0)
            return false;

        float hill = this->hill.sample(
            (float) (x/4) * 1.0, 0.0,
            (float) (z/4) * 1.0
        );

        return hill < -512.0;
    }

    Gpu bool around_spawn(int64_t radius, int64_t step) {
        for (int x=-radius; x<=radius; x+=step) {
            for (int z=-radius; z<=radius; z+=step) {
                if (this->is_monolith(x, z)) {
                    return true;
                }
            }
        }
        return false;
    }
};

/* -------------------------------------------------------------------------- */

__global__ void get_monoliths_world_per_block(
    int start, int seeds,
    float* results
) {
    int idx  = threadIdx.x;
    int dim  = blockDim.x;
    int blk  = blockIdx.x;
    int seed = start + blk;

    __shared__ World world;

    if (threadIdx.x == 0) {
        world.init(seed);

        if (blockIdx.x % 1000 == 0)
            printf("Block %d seed %d\n", blk, seed);

        if (!world.around_spawn(200, 100))
            return;
    }

    __syncthreads();

    int64_t side = 4096;
    int64_t step = 32;
    float   area = 0;

    // Each thread sums its strip
    for (int64_t x=-side+idx; x<=side; x+=step*dim) {
        for (int64_t z=-side; z<=side; z+=step) {
            area += world.is_monolith(x, z) ? step*step : 0.0f;
        }
    }

    atomicAdd(&results[blk], area);
}

__global__ void get_monoliths_world_per_thread(
    int start, int seeds,
    float* results
) {
    int tdx  = threadIdx.x;
    int dim  = blockDim.x;
    int blk  = blockIdx.x;
    int tid  = (blk * dim) + tdx;
    int seed = start + tid;

    World world;
    world.init(seed);

    if (tid % 10000 == 0)
        printf("Block %d seed %d\n", blk, seed);

    if (!world.around_spawn(200, 200))
        return;

    int64_t side = 256;
    int64_t step = 4;

    for (int64_t x=-side; x<=side; x+=step) {
        for (int64_t z=-side; z<=side; z+=step) {
            results[tid] += world.is_monolith(x, z) ? step*step : 0.0f;
        }
    }
}

/* -------------------------------------------------------------------------- */

enum Variant {
    WORLD_PER_THREAD,
    WORLD_PER_BLOCK,
};

int main() {
    int start  = 0;
    int seeds  = 10000000;
    int thread = 64;

    float* d_results;
    hipMalloc(&d_results, seeds * sizeof(float));

    Variant variant = WORLD_PER_THREAD;
    // Variant variant = WORLD_PER_BLOCK;

    if (variant == WORLD_PER_THREAD) {
        int blocks = (seeds + thread - 1) / thread;
        get_monoliths_world_per_thread<<<blocks, thread>>>(start, seeds, d_results);
    } else if (variant == WORLD_PER_BLOCK) {
        int blocks = seeds;
        get_monoliths_world_per_block<<<blocks, thread>>>(start, seeds, d_results);
    }

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Fixme: Make a iterative loop searching chunks of N seeds
    float* results = new float[seeds];
    hipMemcpy(results, d_results, seeds * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_results);

    // Print findings
    for (int i=0; i<seeds; i++) {
        if (results[i] > 0) {
            printf("Seed %llu area: %f\n", start + i, results[i]);
        }
    }

    return 0;
}