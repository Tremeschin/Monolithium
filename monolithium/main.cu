/* -------------------------------------------------------------------------- */
// Failed attempt to find monoliths with cuda.
//
// I'm somewhat sure the lack of memory locality between different seeds and
// the large-ish fractal perlin structs for the tiny cores killed performance.
//
// Maybe it's possible to fix it, or perhaps certain computations are just
// better suited for the CPU even if embarrassingly parallel. This was my first
// time toying with cuda, chances are I did something wrong, PRs are welcome
// to improve this code for the endeavor to find the largest monoliths :^)
//
// Compiling and running:
// - Install CUDA from your package manager (nvcc), have it on path, good luck
//
// Either manually:
//   - Run: `meson setup --buildtype release ./build --reconfigure`
//   - Run: `ninja -C ./build`
//   - Run: `./build/monolithium`
// Or install [uv](https://docs.astral.sh/uv/):
//   - Run: `uv run cuda`
/* -------------------------------------------------------------------------- */

#include <cmath>
#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime.h>

#define Any __global__
#define Gpu __device__
#define Cpu __host__

/* -------------------------------------------------------------------------- */
// Utility functions

Gpu double fade(double t) {
    return t * t * t * (t * (t * 6.0 - 15.0) + 10.0);
}

Gpu double lerp(double t, double a, double b) {
    return a + t * (b - a);
}

Gpu double grad(uint8_t hash, double x, double y, double z) {
    int h = hash & 15;
    double u = h < 8 ? x : y;
    double v = h < 4 ? y : h == 12 || h == 14 ? x : z;
    return ((h & 1) == 0 ? u : -u) + ((h & 2) == 0 ? v : -v);
}

/* -------------------------------------------------------------------------- */
// Java RNG implementation

constexpr int64_t M = (1LL << 48) - 1;
constexpr int64_t A = 0x5DEECE66DLL;
constexpr int64_t C = 11LL;

constexpr double F64_DIV = (1ULL << 53);

struct JavaRNG {
    int64_t state;

    Gpu JavaRNG(uint64_t seed) {
        this->state = ((int64_t) seed ^ A) & M;
    }

    Gpu void step() {
        this->state = (this->state * A + C) & M;
    }

    Gpu int32_t next(uint8_t bits) {
        this->step();
        return ((uint64_t) this->state >> (48 - bits));
    }

    Gpu int32_t next_i32_bound(int32_t max) {
        if (__popc(max) == 1) {
            return (int32_t)(((int64_t) max * (int64_t) this->next(31)) >> 31);
        }
        int32_t next = this->next(31);
        int32_t take = next % max;
        while (next - take + max - 1 < 0) {
            next = this->next(31);
            take = next % max;
        }
        return take;
    }

    Gpu double next_f64() {
        int64_t high = ((int64_t) this->next(26)) << 27;
        int64_t low  =  (int64_t) this->next(27);
        return (double)(high + low) / F64_DIV;
    }
};

/* -------------------------------------------------------------------------- */

struct PerlinNoise {
    uint8_t map[512];
    double xoff;
    double yoff;
    double zoff;

    Gpu void init(JavaRNG& rng) {
        this->xoff = rng.next_f64() * 256.0;
        this->yoff = rng.next_f64() * 256.0;
        this->zoff = rng.next_f64() * 256.0;

        // Start a new 'arange' array
        for (int i=0; i<512; i++) {
            this->map[i] = i & 0xFF;
        }

        // Shuffle the first half
        for (int a=0; a<256; a++) {
            int b = a + rng.next_i32_bound(256 - a);
            uint8_t temp = this->map[a];
            this->map[a] = this->map[b];
            this->map[b] = temp;
        }

        // Mirror to the second half
        for (int i=0; i<256; i++) {
            this->map[i + 256] = this->map[i];
        }
    }

    /// Sample the noise at a given coordinate
    /// - Note: For monoliths, y is often 0.0
    Gpu double sample(double x, double y, double z) {
        x += this->xoff;
        y += this->yoff;
        z += this->zoff;

        // Convert to grid coordinates (512 length)
        int xi = ((int) floor(x)) & 0xFF;
        int yi = ((int) floor(y)) & 0xFF;
        int zi = ((int) floor(z)) & 0xFF;

        // Get the fractional parts
        double xf = x - floor(x);
        double yf = y - floor(y);
        double zf = z - floor(z);

        // Smoothstep-like factors
        double u = fade(xf);
        double v = fade(yf);
        double w = fade(zf);

        // Get the hash values for the corners
        int a  = this->map[xi + 0 + 0] + yi;
        int aa = this->map[yi + a + 0] + zi;
        int ab = this->map[yi + a + 1] + zi;
        int b  = this->map[xi + 0 + 1] + yi;
        int ba = this->map[yi + b + 0] + zi;
        int bb = this->map[yi + b + 1] + zi;

        return lerp(w,
            lerp(v,
                lerp(u, grad(this->map[aa], xf, yf, zf),
                        grad(this->map[ba], xf - 1.0, yf, zf)),
                lerp(u, grad(this->map[ab], xf, yf - 1.0, zf),
                        grad(this->map[bb], xf - 1.0, yf - 1.0, zf))),
            lerp(v,
                lerp(u, grad(this->map[aa + 1], xf, yf, zf - 1.0),
                        grad(this->map[ba + 1], xf - 1.0, yf, zf - 1.0)),
                lerp(u, grad(this->map[ab + 1], xf, yf - 1.0, zf - 1.0),
                        grad(this->map[bb + 1], xf - 1.0, yf - 1.0, zf - 1.0))));
    }

    /// Roll the generator state that would have created a PerlinNoise
    /// - Fast way around without as many memory operations
    Gpu static void discard(JavaRNG& rng, int count) {
        for (int i=0; i<count; i++) {

            // Coordinates f64 offsets
            for (int j=0; j<3; j++) {
                rng.step();
                rng.step();
            }

            // Permutations swapping
            for (int max=256; max>=1; max--) {
                if (__popc(max) == 1) {
                    rng.step();
                } else {
                    int32_t next = rng.next(31);
                    int32_t take = next % max;
                    while (next - take + max - 1 < 0) {
                        next = rng.next(31);
                        take = next % max;
                    }
                }
            }
        }
    }
};

/* -------------------------------------------------------------------------- */

template<int OCTAVES> struct FractalPerlin {
    PerlinNoise noise[OCTAVES];

    Gpu void init(JavaRNG& rng) {
        for (int i=0; i<OCTAVES; i++) {
            this->noise[i].init(rng);
        }
    }

    Gpu double sample(double x, double y, double z) {
        double sum = 0.0;
        for (int i=0; i<OCTAVES; i++) {
            int j = OCTAVES - 1 - i;
            double s = (double)(1 << j);
            sum += this->noise[j].sample(x/s, y/s, z/s) * s;
        }
        return sum;
    }
};

/* -------------------------------------------------------------------------- */

struct World {
    FractalPerlin<10> hill;
    FractalPerlin<16> depth;

    Gpu void init(uint64_t seed) {
        JavaRNG rng(seed);

        // Skip 48 generators priorly used elsewhere
        PerlinNoise::discard(rng, 48);

        this->hill.init(rng);
        this->depth.init(rng);
    }

    // Check if a given coordinate is part of a monolith
    Gpu bool is_monolith(int64_t x, int64_t z) {
        double depth = this->depth.sample(
            (double) (x/4) * 100.0, 0.0,
            (double) (z/4) * 100.0
        );

        if (fabs(depth) < 8000.0)
            return false;

        double hill = this->hill.sample(
            (double) (x/4) * 1.0, 0.0,
            (double) (z/4) * 1.0
        );

        return hill < -512.0;
    }

    // The idea is to get the total area within a region, don't care
    // for position or overcounting, that's a CPU post filter duty
    Gpu int64_t count_monoliths() {
        int64_t count = 0;
        for (int64_t x=-200; x<=200; x+=4) {
            for (int64_t z=-200; z<=200; z+=4) {
                count += this->is_monolith(x, z) ? 16 : 0;
            }
        }
        return count;
    }
};

/* -------------------------------------------------------------------------- */

Any void detect_monoliths(uint64_t start, int64_t* results, int seeds) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= seeds) return;

    printf("Seed %llu\n", start + idx);

    World world;
    world.init(start + idx);
    results[idx] = world.count_monoliths();
}

/* -------------------------------------------------------------------------- */

int main() {
    const int seeds = 500000;
    uint64_t  start = 0;

    int64_t* d_results;
    hipMalloc(&d_results, seeds * sizeof(int64_t));

    int thread = 128;
    int blocks = (seeds + thread - 1) / thread;

    printf("Launching %d blocks × %d threads = %d total threads\n",
           blocks, thread, blocks * thread);

    detect_monoliths<<<blocks, thread>>>(start, d_results, seeds);
    hipDeviceSynchronize();
    printf("Done\n");

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Fixme: Make a iterative loop searching chunks of N seeds
    int64_t* results = new int64_t[seeds];
    hipMemcpy(results, d_results, seeds * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipFree(d_results);

    // Fixme: All areas are zero
    for (int i=0; i<seeds; i++) {
        if (results[i] > 0) {
            printf("Seed %llu area: %lld\n", start + i, results[i]);
        }
    }

    delete[] results;
    return 0;
}